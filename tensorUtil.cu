#include "hip/hip_runtime.h"
#include "tensorUtil.h"
#include "tensorCuda.h"

static void assertTensor(Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

static void assertShapeEqual(int ndim1, int *dims1, int ndim2, int *dims2)
{
     assert(ndim1 == ndim2);
     while (--ndim1 >= 0)
          assert(dims1[ndim1] == dims2[ndim1]);
}

void *cloneMem(void *src, size_t size, const CloneKind kind)
{
     assert(src && kind);
     void *p;
     switch (kind) {
     case H2H:
          p = malloc(size);
          assert(p);
          memmove(p, src, size);
          return p;
          break;
     case H2D:
          hipMalloc(&p, size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyHostToDevice);
          return p;
          break;
     case D2D:
          hipMalloc(&p, size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyDeviceToDevice);
          return p;
          break;
     case D2H:
          p = malloc(size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyDeviceToHost);
          return p;
          break;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }

}

int computeLength(int ndim, int *dims)
{
     assert(dims);
     int i, len = 1;
     for (i = 0; i < ndim; i++)
          len *= dims[i];
     return len;
}

Tensor *createTensor(float *data, int ndim, int *dims)
{
     Tensor *t = (Tensor *)malloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)malloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

void printTensor(Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     float *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          printf("%s", right_buf);
          if (*right_buf != '\0') {
               putchar('\n');
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    putchar(' ');
          }
          printf("%s", left_buf);
          if (*left_buf == '\0')
               putchar(' ');
          printf(fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          putchar(']');
     putchar('\n');
}

Tensor *sliceTensor(Tensor *src, int dim, int start, int len)
{
     assertTensor(src);
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)malloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)malloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     dst->data = (float *)malloc(dst->len * sizeof(float));

     int i, block_size, block_num; /* block size and number for copy operation */
     for (i = dim+1, block_size = 1; i < dst->ndim; i++)
          block_size *= dst->dims[i];
     for (i = 0, block_num = 1; i <= dim; i++)
          block_num *= dst->dims[i];

     int index;
     float *dp = dst->data, *sp = src->data;
     size_t floats_size = block_size * sizeof(float);
     for (i = 0; i < block_num; i++) {
          index = i / len * src->dims[dim] + i % len + start;
          memmove(dp+i*block_size, sp+index*block_size, floats_size);
     }

     return dst;
}

void *sliceTensorCuda(Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assertTensor(src);
     assertTensor(dst);
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     /* Tensor *dst = (Tensor *)malloc(sizeof(Tensor)); /\* new tensor *\/ */
     /* dst->ndim = src->ndim; */
     /* dst->dims = (int *)malloc(sizeof(int) * dst->ndim); */
     /* memmove(dst->dims, src->dims, sizeof(int) * dst->ndim); */
     /* dst->dims[dim] = len; */
     /* dst->len = src->len / src->dims[dim] * len; */
     /* hipMalloc(&dst->data, sizeof(float) * dst->len); */

     int i, block_size, block_num; /* block size and number of cuda threads */
     int ddim = dst->dims[dim], sdim = src->dims[dim];
     for (i = dim+1, block_size = 1; i < dst->ndim; i++)
          block_size *= dst->dims[i];
     for (i = 0, block_num = 1; i <= dim; i++)
          block_num *= dst->dims[i];

     sliceTensorKernel<<<block_num, block_size>>>(dst->data, src->data, ddim, sdim, start, block_size);

     return dst;
}

/* in-place reshape tensor */
Tensor *reshapeTensor(Tensor *src, int newNdim, int *newDims)
{
     assertTensor(src);
     assert(newDims);
     assert(src->len == computeLength(newNdim, newDims));
     Tensor *dst = createTensor(src->data, newNdim, newDims); /* new tensor */
     return dst;
}

/* current only support dim = src->dims[src->ndim-1] */
void reduceArgMax(Tensor *src, Tensor **dst, Tensor **arg, int dim)
{
     clock_t start, end;

     assertTensor(src);
     assert(dim < src->ndim);
     assert(dim == src->ndim-1); /* TODO: get rid of this limit */

     Tensor *dstp = (Tensor *)malloc(sizeof(Tensor));
     dstp->ndim = src->ndim;
     dstp->dims = (int *)malloc(sizeof(int) * dstp->ndim);
     memmove(dstp->dims, src->dims, sizeof(int) * dstp->ndim);
     dstp->dims[dim] = 1;
     dstp->len = computeLength(dstp->ndim, dstp->dims);
     hipMalloc(&dstp->data, sizeof(float) * dstp->len);

     Tensor *argp = (Tensor *)malloc(sizeof(Tensor));
     argp->ndim = dstp->ndim;
     argp->dims = (int *)malloc(sizeof(int) * argp->ndim);
     memmove(argp->dims, dstp->dims, sizeof(int) * argp->ndim);
     argp->len = dstp->len;

     start = clock();
     hipMalloc(&argp->data, sizeof(float) * argp->len);
     end = clock();
     printf("alloc in %ld\n", end - start);

     int i, thread_num, block_size, block_num;
     for (i = 0, thread_num = 1; i < dim; i++)
          thread_num *= dstp->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     start = clock();
     reduceArgMaxKernel<<<block_num, block_size>>>(src->data, dstp->data, argp->data, src->dims[dim], block_size);
     end = clock();
     printf("kernel in %ld\n", end - start);

     *dst = dstp;
     *arg = argp;
}

/* Tensor *multiplyElement(Tensor *src1, Tensor *src2) */
/* { */
/*      assertTensor(src1); */
/*      assertTensor(src2); */
/*      assertShapeEqual(src1->ndim, src1->dims, src2->ndim, src2->dims); */

/*      Tensor *dst = (Tensor *)malloc(sizeof(Tensor)); */
/*      dst->ndim = src1->ndim; */
/*      dst->dims = (int *)malloc(sizeof(int) dst->ndim); */
/*      memmove(dst->dims, src1->dims, sizeof(int) * dst->ndim); */
/*      dst->len = src1->len; */
/*      hipMalloc(&dst->data, sizeof(float) * dst->len); */


/* } */
