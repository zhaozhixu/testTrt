#include "tensorUtil.h"
#include "tensorCuda.h"

static void assertTensor(const Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isShapeEqual(int ndim1, const int *dims1, int ndim2, const int *dims2)
{
     if (ndim1 == ndim2) {
          while (--ndim1 >= 0)
               if (dims1[ndim1] != dims2[ndim1])
                    return 0;
          return 1;
     }
     return 0;
}

void *cloneMem(const void *src, size_t size, CloneKind kind)
{
     assert(src && kind);
     void *p;
     switch (kind) {
     case H2H:
          p = malloc(size);
          assert(p);
          memmove(p, src, size);
          return p;
     case H2D:
          hipMalloc(&p, size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyHostToDevice);
          return p;
     case D2D:
          hipMalloc(&p, size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyDeviceToDevice);
          return p;
     case D2H:
          p = malloc(size);
          assert(p);
          hipMemcpy(p, src, size, hipMemcpyDeviceToHost);
          return p;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }

}

int computeLength(int ndim, const int *dims)
{
     assert(dims);
     int i, len = 1;
     for (i = 0; i < ndim; i++)
          len *= dims[i];
     return len;
}

Tensor *createTensor(float *data, int ndim, const int *dims)
{
     Tensor *t = (Tensor *)malloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)malloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

void printTensor(const Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     float *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          printf("%s", right_buf);
          if (*right_buf != '\0') {
               putchar('\n');
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    putchar(' ');
          }
          printf("%s", left_buf);
          if (*left_buf == '\0')
               putchar(' ');
          printf(fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          putchar(']');
     putchar('\n');
}

Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len)
{
     assertTensor(src);
     assert(dim <= src->ndim && dim >= 0);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)malloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)malloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     dst->data = (float *)malloc(dst->len * sizeof(float));
     return dst;
}

Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assertTensor(src);
     assertTensor(dst);
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     int i, block_size, block_num; /* block size and number for copy operation */
     for (i = dim+1, block_size = 1; i < dst->ndim; i++)
          block_size *= dst->dims[i];
     for (i = 0, block_num = 1; i <= dim; i++)
          block_num *= dst->dims[i];

     int index;
     float *dp = dst->data, *sp = src->data;
     size_t floats_size = block_size * sizeof(float);
     for (i = 0; i < block_num; i++) {
          index = i / len * src->dims[dim] + i % len + start;
          memmove(dp+i*block_size, sp+index*block_size, floats_size);
     }

     return dst;
}

Tensor *creatSlicedTensorCuda(const Tensor *src, int dim, int start, int len)
{
     assertTensor(src);
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)malloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)malloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     hipMalloc(&dst->data, sizeof(float) * dst->len);
     return dst;
}

void *sliceTensorCuda(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assertTensor(src);
     assertTensor(dst);
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     int i, block_size, block_num; /* block size and number of cuda threads */
     int ddim = dst->dims[dim], sdim = src->dims[dim];
     for (i = dim+1, block_size = 1; i < dst->ndim; i++)
          block_size *= dst->dims[i];
     for (i = 0, block_num = 1; i <= dim; i++)
          block_num *= dst->dims[i];

     sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, sdim, ddim, start, block_size);

     return dst;
}

/* in-place reshape tensor */
Tensor *reshapeTensor(const Tensor *src, int newNdim, const int *newDims)
{
     assertTensor(src);
     assert(newDims);
     assert(src->len == computeLength(newNdim, newDims));
     Tensor *dst = createTensor(src->data, newNdim, newDims); /* new tensor */
     return dst;
}

/* current only support dim = src->dims[src->ndim-1] */
Tensor *createReducedTensor(const Tensor *src, int dim)
{
     assertTensor(src);
     assert(dim < src->ndim && dim >= 0);
     assert(dim == src->ndim-1); /* TODO: get rid of this limit */

     Tensor *dst = (Tensor *)malloc(sizeof(Tensor));
     dst->ndim = src->ndim;
     dst->dims = (int *)malloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = 1;
     dst->len = computeLength(dst->ndim, dst->dims);
     hipMalloc(&dst->data, sizeof(float) * dst->len);
     return dst;
}

/* current only support dim = src->dims[src->ndim-1] */
void *reduceArgMax(const Tensor *src, Tensor *dst, Tensor *arg, int dim)
{
     assertTensor(src);
     assertTensor(dst);
     assertTensor(arg);
     assert(dim == src->ndim-1); /* TODO: get rid of this limit */
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == 1 : dst->dims[i] == src->dims[i] &&
                 i == dim ? arg->dims[i] == 1 : arg->dims[i] == src->dims[i]);

     int i, thread_num, block_size, block_num;
     for (i = 0, thread_num = 1; i < dim; i++)
          thread_num *= dst->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     reduceArgMaxKernel<<<block_num, block_size>>>(src->data, dst->data, arg->data, src->dims[dim], block_size);

     return dst;
}

Tensor *multiplyElement(const Tensor *src1, const Tensor *src2, Tensor *dst)
{
     assertTensor(src1);
     assertTensor(src2);
     assertTensor(dst);
     assert(isShapeEqual(src1->ndim, src1->dims, src2->ndim, src2->dims));
     assert(isShapeEqual(src1->ndim, src1->dims, dst->ndim, dst->dims));

     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     multiplyElementKernel<<<block_num, block_size>>>(src1->data, src2->data, dst->data, block_size);

     return dst;
}
